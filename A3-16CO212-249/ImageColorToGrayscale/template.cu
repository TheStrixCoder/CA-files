#include "hip/hip_runtime.h"
#include"wb.h"
#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>

//@@ define error checking macro here.
#define errCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      printErrorLog(ERROR, "Failed to run stmt ", #stmt);                         \
      printErrorLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

//@@ INSERT CODE HERE

void __global__ RGBToGray(float* devIn, float* devOut, int imgWd, int imgHt)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;

  if(idx>=0 && idy>=0 && idx < imgHt && idy < imgWd)
  {
    int id = idx * imgWd + idy;
    
    devOut[id] = 0.21*devIn[3*id] + 0.71*devIn[3*id+1] + 0.07*devIn[3*id+2];
  }
}

#define THREAD_NUM 16

int main(int argc, char *argv[]) {

  int imageChannels;
  int imageWidth;
  int imageHeight;
  char *inputImageFile;
  wbImage_t inputImage;
  wbImage_t outputImage;
  float *hostInputImageData;
  float *hostOutputImageData;
  float *deviceInputImageData;
  float *deviceOutputImageData;

  /* parse the input arguments */
  //@@ Insert code here

  if(argc != 9)
  {
    printf("Usage:  ./TiledMatrixMultiplication_Template -e <expected.pbm> -i <input.ppm> -o <output.pbm> -t matrix");
    exit(0);
  }

  wbArg_t args = {argc, argv};

  inputImageFile = wbArg_getInputFile(args, 3);

  inputImage = wbImport(inputImageFile);

  imageWidth  = wbImage_getWidth(inputImage);
  
  imageHeight = wbImage_getHeight(inputImage);
  // For this lab the value is always 3
  imageChannels = wbImage_getChannels(inputImage);

  // Since the image is monochromatic, it only contains one channel
  outputImage = wbImage_new(imageWidth, imageHeight, 1);

  hostInputImageData  = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  hipMalloc((void **)&deviceInputImageData,
             imageWidth * imageHeight * imageChannels * sizeof(float));
  hipMalloc((void **)&deviceOutputImageData,
             imageWidth * imageHeight * sizeof(float));
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  hipMemcpy(deviceInputImageData, hostInputImageData,
             imageWidth * imageHeight * imageChannels * sizeof(float),
             hipMemcpyHostToDevice);
  wbTime_stop(Copy, "Copying data to the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Compute, "Doing the computation on the GPU");
  //@@ INSERT CODE HERE

  dim3 blockSize(THREAD_NUM, THREAD_NUM, 1);
  dim3 gridSize((int)ceil(imageWidth/(float)blockSize.x), (int)ceil(imageHeight/(float)blockSize.y), 1);

  RGBToGray<<<gridSize, blockSize>>>(deviceInputImageData, deviceOutputImageData, imageWidth, imageHeight);
  
  wbTime_stop(Compute, "Doing the computation on the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Copy, "Copying data from the GPU");
  hipMemcpy(hostOutputImageData, deviceOutputImageData,
             imageWidth * imageHeight * sizeof(float),
             hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  wbSolution(args, 5, outputImage);

  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);

  wbImage_delete(outputImage);
  wbImage_delete(inputImage);

  return 0;
}
