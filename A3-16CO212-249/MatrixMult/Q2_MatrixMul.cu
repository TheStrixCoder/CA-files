#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>


//Number of threads in each dimension of the block.
#define THREAD_NUM 16

// CUDA kernel
__global__ void matrixMul(int *A, int *B, int *C, int n)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	int num = n;

    	if (row < num && col < num)
    	{
        	long Cvalue = 0;
		for (int i = 0; i < num; i++)
		{
			Cvalue += A[row * num + i] * B[i * num + col];
		}
		C[row * num + col] = Cvalue;
    	}
}


// Main
int main(void)
{
    	// Error code to check return values for CUDA calls
    	hipError_t err = hipSuccess;

    	int num = 512, i, j;
    	size_t size = num * num * sizeof(int);
    	printf("\n\tMatrix multiplication of two %d * %d matrices\n\n", num, num);

    	int h_A[num][num], h_B[num][num], h_C[num][num];
	
	printf("Initializing host input vectors...\n");
    	for (int i = 0; i < num; i++)
    	{
		for (int j = 0; j < num; j++)
        	{
			
			h_A[i][j] = i*j;
			h_B[i][j] = i+1;
		}
    	}

    	
	printf("Allocating device memory...\n");
    	int *d_A = NULL;
    	err = hipMalloc((void **)&d_A, size);
	if (err != hipSuccess)
    	{
        	fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}

    	int *d_B = NULL;
    	err = hipMalloc((void **)&d_B, size);
	if (err != hipSuccess)
    	{
        	fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}

    	int *d_C = NULL;
    	err = hipMalloc((void **)&d_C, size);

    	if (err != hipSuccess)
    	{
        	fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}

	printf("Copying input from host to device...\n");
    	err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	if (err != hipSuccess)
    	{
        	fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}

	err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	if (err != hipSuccess)
    	{
        	fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}
	printf("Input matrices...\n\nMatrix A: \n");

        for(i=0; i<num; i++){
                for(j=0; j<num; j++)
                        printf("%d ", h_A[i][j]);
                printf("\n");
        }
        printf("\nMatrix B: ");

        for(i=0; i<num; i++){
                for(j=0; j<num; j++)
                        printf("%d ", h_B[i][j]);
                printf("\n");
        }
        printf("\n");

    	// Launch CUDA Kernel
	printf("vector multiplication kernel...\n");
	dim3 dimBlock(THREAD_NUM, THREAD_NUM, 1);
    	dim3 dimGrid((int) ceil((float)num/dimBlock.x), (int) ceil((float)num/dimBlock.y), 1);
    	matrixMul<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, num);
   	err = hipGetLastError();

    	if (err != hipSuccess)
    	{
        	fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}

    	// Copy result from device to host
	printf("Copying result from device to host...\n");
    	err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess)
    	{
        	fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}

        printf("Displaying the output matrix...\n\nMatrix C: \n");

	for(i=0; i<num; i++){
		for(j=0; j<num; j++)
			printf("%d ", h_C[i][j]);
		printf("\n");
	}
	printf("\n");

	// Free device global memory
	printf("Freeing device memory...\n");
    	err = hipFree(d_A);
	

    	err = hipFree(d_B);
	
    	err = hipFree(d_C);

    	printf("Done.\n\n");
    	return 0;
}
